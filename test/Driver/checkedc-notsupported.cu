
#include <hip/hip_runtime.h>
// Checked C extension is not supported for CUDA.   Make sure driver
// rejects the flag.
//
// RUN: not %clang -fcheckedc-extension %s 2>&1 | FileCheck %s
// CHECK: error: invalid argument '-fcheckedc-extension' not allowed with 'CUDA'

extern void f() {}


